#include "hip/hip_runtime.h"
#include "Emulate.h"
#include "Wrapper.h"
#include <stdio.h>
#include <vector>
#include <thread>

#if ENABLE_CUDA

#include <hip/hip_runtime.h>
#include <>

namespace CUDA_TEST {
	CUDA_Emulate::CUDA_Emulate(unsigned int countBlock, unsigned int countThread, std::function<void(CUDA_TEST::Index)> fun) {
		std::vector<std::thread> threads;
		threads.reserve(countBlock * countThread);

		for (unsigned int iThread = 0; iThread < countThread; ++iThread) {
			for (unsigned int iBlock = 0; iBlock < countBlock; ++iBlock) {
				threads.emplace_back([iThread, iBlock, countThread, fun]() {
					fun(CUDA_TEST::Index(iBlock, iThread, countThread));
				});
			}
		}

		for (std::thread& th : threads) {
			th.join();
		}
	}

	unsigned int CUDA_Emulate::hipMemcpyHostToDevice = 1;
	unsigned int CUDA_Emulate::hipMemcpyDeviceToHost = 2;
}

/// CPU //////////////////////////////////////////////////////////////////////////////

namespace {
	void UpdateCPU(unsigned int* count, unsigned int* offset, CUDA::Vector3* positions, CUDA_TEST::Index indexData) {
		int indexT = indexData.threadIdx.x + indexData.blockIdx.x * indexData.blockDim.x;
		int startIndex = indexT * *offset;
		int countIndex = startIndex + *offset;
		if (countIndex >= *count) {
			countIndex = *count;
		}
		//printf("\nUpdateCpu INDEXES [%i, %i] [%i, %i, %i]\n", startIndex, countIndex, indexData.threadIdx.x, indexData.blockIdx.x, indexData.blockDim.x);

		for (int index = startIndex; index < countIndex; ++index) {
			CUDA::Vector3& pos = positions[index];
			//printf("          POS: [%i] [%f, %f, %f] => ", index, pos.x, pos.y, pos.z);
			pos.x += pos.x;
			pos.y += pos.y;
			pos.z += pos.z;
			//printf(" [%f, %f, %f]\n", pos.x, pos.y, pos.z);
		}
	}
}

/// GPU //////////////////////////////////////////////////////////////////////////////

namespace {
	__global__ void UpdateGPU(unsigned int* count, unsigned int* offset, CUDA::Vector3* positions) {
		int indexT = threadIdx.x + blockIdx.x * blockDim.x;
		int startIndex = indexT * *offset;
		int countIndex = startIndex + *offset;
		if (countIndex >= *count) {
			countIndex = *count;
		}
		//printf("\nUpdateGpu INDEXES [%i, %i] [%i, %i, %i]\n", startIndex, countIndex, threadIdx.x, blockIdx.x, blockDim.x);

		for (int index = startIndex; index < countIndex; ++index) {
			CUDA::Vector3& pos = positions[index];
			//printf("          POS: [%i] [%f, %f, %f] => ", index, pos.x, pos.y, pos.z);
			pos.x += pos.x;
			pos.y += pos.y;
			pos.z += pos.z;
			//printf(" [%f, %f, %f]\n", pos.x, pos.y, pos.z);
		}
	}
}

/////////////////////////////////////////////////////////////////////////////////

void CUDA_TEST::Test() {
	unsigned int count = 100;
	
	std::vector<float> masses;
	masses.reserve(count);
	std::vector<CUDA::Vector3> positionsCpu;
	positionsCpu.reserve(count);
	std::vector<CUDA::Vector3> positionsGpu;
	positionsGpu.reserve(count);

	for (int i = (count-1); i >= 0; --i) {
		positionsCpu.emplace_back((float)i, (float)i, (float)i);
		positionsGpu.emplace_back((float)i, (float)i, (float)i);
		masses.emplace_back((float)i + 1.f);
	}

	// CPU
	{
		std::vector<CUDA::Vector3>& positions = positionsCpu;
		std::vector<CUDA::Vector3> forces;
		forces.resize(count);

		const unsigned int maxCountBlock = 16;
		const unsigned int maxCountThread = 16;

		unsigned int countBlock;
		unsigned int countThread;
		unsigned int offset;

		CUDA::GetOffsets(count, maxCountBlock, maxCountThread, countBlock, countThread, offset);
		printf("\nCPU BEGIN count: %i, offset %i, countBlock %i, countThread %i\n", count, offset, countBlock, countThread);

		unsigned int* devCount;
		unsigned int* devOffset;
		CUDA::Vector3* devPositions;
		float* devMasses;
		CUDA::Vector3* devForces;

		CUDA_TEST::CUDA_Emulate::hipMalloc(&devCount, sizeof(unsigned int));
		CUDA_TEST::CUDA_Emulate::hipMalloc(&devOffset, sizeof(unsigned int));
		CUDA_TEST::CUDA_Emulate::hipMalloc(&devPositions, count * sizeof(CUDA::Vector3));
		CUDA_TEST::CUDA_Emulate::hipMalloc(&devMasses, count * sizeof(float));
		CUDA_TEST::CUDA_Emulate::hipMalloc(&devForces, count * sizeof(CUDA::Vector3));

		CUDA_TEST::CUDA_Emulate::hipMemcpy(devCount, &count, sizeof(unsigned int), CUDA_TEST::CUDA_Emulate::hipMemcpyHostToDevice);
		CUDA_TEST::CUDA_Emulate::hipMemcpy(devOffset, &offset, sizeof(unsigned int), CUDA_TEST::CUDA_Emulate::hipMemcpyHostToDevice);
		CUDA_TEST::CUDA_Emulate::hipMemcpy(devPositions, positions.data(), count * sizeof(CUDA::Vector3), CUDA_TEST::CUDA_Emulate::hipMemcpyHostToDevice);
		CUDA_TEST::CUDA_Emulate::hipMemcpy(devMasses, masses.data(), count * sizeof(float), CUDA_TEST::CUDA_Emulate::hipMemcpyHostToDevice);
		CUDA_TEST::CUDA_Emulate::hipMemcpy(devForces, forces.data(), count * sizeof(CUDA::Vector3), CUDA_TEST::CUDA_Emulate::hipMemcpyHostToDevice);

		CUDA_Emulate(countBlock, countThread, [devCount, devOffset, devPositions](Index indexData) {
			UpdateCPU(devCount, devOffset, devPositions, indexData);
		});

		CUDA_TEST::CUDA_Emulate::hipMemcpy(&count, devCount, sizeof(unsigned int), CUDA_TEST::CUDA_Emulate::hipMemcpyDeviceToHost);
		CUDA_TEST::CUDA_Emulate::hipMemcpy(&offset, devOffset, sizeof(unsigned int), CUDA_TEST::CUDA_Emulate::hipMemcpyDeviceToHost);
		CUDA_TEST::CUDA_Emulate::hipMemcpy(positions.data(), devPositions, count * sizeof(CUDA::Vector3), CUDA_TEST::CUDA_Emulate::hipMemcpyDeviceToHost);
		CUDA_TEST::CUDA_Emulate::hipMemcpy(masses.data(), devMasses, count * sizeof(float), CUDA_TEST::CUDA_Emulate::hipMemcpyDeviceToHost);
		CUDA_TEST::CUDA_Emulate::hipMemcpy(forces.data(), devForces, count * sizeof(CUDA::Vector3), CUDA_TEST::CUDA_Emulate::hipMemcpyDeviceToHost);

		CUDA_TEST::CUDA_Emulate::hipFree(devCount);
		CUDA_TEST::CUDA_Emulate::hipFree(devOffset);
		CUDA_TEST::CUDA_Emulate::hipFree(devPositions);
		CUDA_TEST::CUDA_Emulate::hipFree(devMasses);
		CUDA_TEST::CUDA_Emulate::hipFree(devForces);

		printf("\nCPU END \n");
	}

	// GPU
	{
		std::vector<CUDA::Vector3>& positions = positionsGpu;
		std::vector<CUDA::Vector3> forces;
		forces.resize(count);

		const unsigned int maxCountBlock = 16;
		const unsigned int maxCountThread = 16;

		unsigned int countBlock;
		unsigned int countThread;
		unsigned int offset;

		CUDA::GetOffsets(count, maxCountBlock, maxCountThread, countBlock, countThread, offset);
		printf("\nGPU BEGIN count: %i, offset %i, countBlock %i, countThread %i\n", count, offset, countBlock, countThread);

		unsigned int* devCount;
		unsigned int* devOffset;
		CUDA::Vector3* devPositions;
		float* devMasses;
		CUDA::Vector3* devForces;

		hipMalloc(&devCount, sizeof(unsigned int));
		hipMalloc(&devOffset, sizeof(unsigned int));
		hipMalloc(&devPositions, count * sizeof(CUDA::Vector3));
		hipMalloc(&devMasses, count * sizeof(float));
		hipMalloc(&devForces, count * sizeof(CUDA::Vector3));

		hipMemcpy(devCount, &count, sizeof(unsigned int), hipMemcpyHostToDevice);
		hipMemcpy(devOffset, &offset, sizeof(unsigned int), hipMemcpyHostToDevice);
		hipMemcpy(devPositions, positions.data(), count * sizeof(CUDA::Vector3), hipMemcpyHostToDevice);
		hipMemcpy(devMasses, masses.data(), count * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(devForces, forces.data(), count * sizeof(CUDA::Vector3), hipMemcpyHostToDevice);

		UpdateGPU << <countBlock, countThread >> > (devCount, devOffset, devPositions);

		hipMemcpy(&count, devCount, sizeof(unsigned int), hipMemcpyDeviceToHost);
		hipMemcpy(&offset, devOffset, sizeof(unsigned int), hipMemcpyDeviceToHost);
		hipMemcpy(positions.data(), devPositions, count * sizeof(CUDA::Vector3), hipMemcpyDeviceToHost);
		hipMemcpy(masses.data(), devMasses, count * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(forces.data(), devForces, count * sizeof(CUDA::Vector3), hipMemcpyDeviceToHost);

		hipFree(devCount);
		hipFree(devOffset);
		hipFree(devPositions);
		hipFree(devMasses);
		hipFree(devForces);

		printf("\nGPU END \n");
	}

	bool equal = true;
	bool printData = true;;
	{
		for (size_t i = 0; i < count; ++i) {
			if (positionsCpu[i].x == positionsGpu[i].x && positionsCpu[i].y == positionsGpu[i].y && positionsCpu[i].z == positionsGpu[i].z) {
				if (printData) {
					printf("\tPOS[%i]:\n[%f, %f, %f] !=\n[%f, %f, %f]\n", i,
						positionsCpu[i].x, positionsCpu[i].y, positionsCpu[i].z,
						positionsGpu[i].x, positionsGpu[i].y, positionsGpu[i].z);
				}
			} else {
				equal = false;

				if (printData) {
					printf("\tPOS[%i]:\n[%f, %f, %f] !=\n[%f, %f, %f] FAIL\n", i,
						positionsCpu[i].x, positionsCpu[i].y, positionsCpu[i].z,
						positionsGpu[i].x, positionsGpu[i].y, positionsGpu[i].z);
				} else {
					break;
				}
			}
		}
	}

	if (equal) {
		printf("\nCOMPARE OK \n...............................................\n");
	} else {
		printf("\nCOMPARE FAIL \n...............................................\n");
	}	
}

#else

void CUDA_TEST::Test() { }

#endif
