#include "hip/hip_runtime.h"

#include "WrapperX1.h"

bool WrapperX1::sync = true;
int WrapperX1::tag = 0;
int WrapperX1::tagCurrent = -1;

#include <hip/hip_runtime.h>
#include <>

#include <stdio.h>
#include <iostream>
#include <string>
#include <thread>

#include "Emulate.h"

//////////////////////////////////////////////////////////////////////////////////////////////////////////
/// CPU
//////////////////////////////////////////////////////////////////////////////////////////////////////////

namespace {
	void CalcForcesCpu(unsigned int* count, unsigned int* offset, cuda::Vector3* positions, float* masses, cuda::Vector3* forces, CUDA_TEST::Index indexData) {
		int indexT = indexData.threadIdx.x + indexData.blockIdx.x * indexData.blockDim.x;
		int startIndex = indexT * *offset;
		int countIndex = startIndex + *offset;
		if (countIndex >= *count) {
			countIndex = *count;
		}

		float constGravity = 0.01;
		float gravityX = 0.0;
		float gravityY = 0.0;
		float gravityZ = 0.0;
		float dist = 0.0;
		float force = 0.0;

		for (int index = startIndex; index < countIndex; ++index) {
			cuda::Vector3* pos = &positions[index];
			float mass = masses[index];
			forces[index].x = 0.f;
			forces[index].y = 0.f;
			forces[index].z = 0.f;

			for (size_t otherIndex = 0; otherIndex < *count; ++otherIndex) {
				if (index == otherIndex) {
					continue;
				}

				gravityX = positions[otherIndex].x - positions[index].x;
				gravityY = positions[otherIndex].y - positions[index].y;
				gravityZ = positions[otherIndex].z - positions[index].z;

				dist = sqrt(gravityX * gravityX + gravityY * gravityY + gravityZ * gravityZ);
				gravityX /= dist;
				gravityY /= dist;
				gravityZ /= dist;

				force = constGravity * (mass * masses[otherIndex]) / (dist * dist);
				gravityX *= force;
				gravityY *= force;
				gravityZ *= force;

				forces[index].x += gravityX;
				forces[index].y += gravityY;
				forces[index].z += gravityZ;
			}
		}
	}

	void UpdatePositionsCpu(unsigned int* count, unsigned int* offset, cuda::Vector3* positions, cuda::Vector3* velocities, float* masses, cuda::Vector3* forces, float* dt, CUDA_TEST::Index indexData) {
		int indexT = indexData.threadIdx.x + indexData.blockIdx.x * indexData.blockDim.x;
		int startIndex = indexT * *offset;
		int sizeIndex = startIndex + *offset;
		if (sizeIndex >= *count) {
			sizeIndex = *count;
		}

		float accelerationX;
		float accelerationY;
		float accelerationZ;
		float appendVelocityX;
		float appendVelocityY;
		float appendVelocityZ;

		for (int index = startIndex; index < sizeIndex; ++index) {
			accelerationX = forces[index].x / masses[index];
			accelerationY = forces[index].y / masses[index];
			accelerationZ = forces[index].z / masses[index];

			appendVelocityX = accelerationX * *dt;
			appendVelocityY = accelerationY * *dt;
			appendVelocityZ = accelerationZ * *dt;

			velocities[index].x += appendVelocityX;
			velocities[index].y += appendVelocityY;
			velocities[index].z += appendVelocityZ;

			positions[index].x += velocities[index].x * *dt;
			positions[index].y += velocities[index].y * *dt;
			positions[index].z += velocities[index].z * *dt;
		}
	}
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////

void WrapperX1::CalculateForceCPU(cuda::Buffer& buffer) {
	const unsigned int maxCountBlock = 1;
	const unsigned int maxCountThread = CUDA::multithread ? std::thread::hardware_concurrency() : 1;

	unsigned int countBlock;
	unsigned int countThread;
	unsigned int offset;
	unsigned int count = buffer.count;

	CUDA::GetOffsets(count, maxCountBlock, maxCountThread, countBlock, countThread, offset);

	unsigned int* devCount = &count;
	unsigned int* devOffset = &offset;
	cuda::Vector3* devPositions = buffer.positions.data();
	float* devMasses = buffer.masses.data();
	cuda::Vector3* devForces = buffer.forces.data();

	if (countThread > 1) {
		CUDA_TEST::CUDA_Emulate(countBlock, countThread, [&](CUDA_TEST::Index indexData) {
			CalcForcesCpu(devCount, devOffset, devPositions, devMasses, devForces, indexData);
		});
	}
	else {
		CUDA_TEST::Index indexData(0, 0, 1);
		CalcForcesCpu(devCount, devOffset, devPositions, devMasses, devForces, indexData);
	}
}

void WrapperX1::UpdatePositionCPU(cuda::Buffer& buffer, float dt) {
	const unsigned int maxCountBlock = 1;
	const unsigned int maxCountThread = CUDA::multithread ? std::thread::hardware_concurrency() : 1;

	unsigned int countBlock;
	unsigned int countThread;
	unsigned int offset;
	unsigned int count = buffer.count;

	CUDA::GetOffsets(count, maxCountBlock, maxCountThread, countBlock, countThread, offset);

	unsigned int* devCount = &count;
	unsigned int* devOffset = &offset;
	cuda::Vector3* devPositions = buffer.positions.data();
	float* devMasses = buffer.masses.data();
	cuda::Vector3* devForces = buffer.forces.data();
	cuda::Vector3* devVelocities = buffer.velocities.data();
	float* devDt = &dt;

	if (countThread > 1) {
		CUDA_TEST::CUDA_Emulate(countBlock, countThread, [&](CUDA_TEST::Index indexData) {
			UpdatePositionsCpu(devCount, devOffset, devPositions, devVelocities, devMasses, devForces, devDt, indexData);
		});
	}
	else {
		CUDA_TEST::Index indexData(0, 0, 1);
		UpdatePositionsCpu(devCount, devOffset, devPositions, devVelocities, devMasses, devForces, devDt, indexData);
	}
}


//////////////////////////////////////////////////////////////////////////////////////////////////////////
// GPU ///////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////

namespace {
	__global__ void CalcForcesGpu(unsigned int* count, unsigned int* offset, CUDA::Vector3* positions, float* masses, CUDA::Vector3* forces) {
		int indexT = threadIdx.x + blockIdx.x * blockDim.x;
		int startIndex = indexT * *offset;
		int countIndex = startIndex + *offset;
		if (countIndex >= *count) {
			countIndex = *count;
		}

		float constGravity = 0.01;
		float gravityX = 0.0;
		float gravityY = 0.0;
		float gravityZ = 0.0;
		float dist = 0.0;
		float force = 0.0;

		for (int index = startIndex; index < countIndex; ++index) {
			CUDA::Vector3* pos = &positions[index];
			float mass = masses[index];
			forces[index].x = 0.f;
			forces[index].y = 0.f;
			forces[index].z = 0.f;

			for (size_t otherIndex = 0; otherIndex < *count; ++otherIndex) {
				if (index == otherIndex) {
					continue;
				}

				gravityX = positions[otherIndex].x - positions[index].x;
				gravityY = positions[otherIndex].y - positions[index].y;
				gravityZ = positions[otherIndex].z - positions[index].z;

				dist = sqrt(gravityX * gravityX + gravityY * gravityY + gravityZ * gravityZ);
				gravityX /= dist;
				gravityY /= dist;
				gravityZ /= dist;

				force = constGravity * (mass * masses[otherIndex]) / (dist * dist);
				gravityX *= force;
				gravityY *= force;
				gravityZ *= force;

				forces[index].x += gravityX;
				forces[index].y += gravityY;
				forces[index].z += gravityZ;

				atomicAdd(&forces[index].x, gravityX);
				atomicAdd(&forces[index].y, gravityY);
				atomicAdd(&forces[index].z, gravityZ);
			}
		}
	}

	__global__ void CalcForcesGpuSync(unsigned int* count, unsigned int* offset, CUDA::Vector3* positions, float* masses, CUDA::Vector3* forces) {
		int indexT = threadIdx.x + blockIdx.x * blockDim.x;
		int startIndex = indexT * *offset;
		int countIndex = startIndex + *offset;
		if (countIndex >= *count) {
			countIndex = *count;
		}

		float constGravity = 0.01;
		float gravityX = 0.0;
		float gravityY = 0.0;
		float gravityZ = 0.0;
		float dist = 0.0;
		float force = 0.0;

		for (int index = startIndex; index < countIndex; ++index) {
			CUDA::Vector3* pos = &positions[index];
			float mass = masses[index];
			forces[index].x = 0.f;
			forces[index].y = 0.f;
			forces[index].z = 0.f;

			for (size_t otherIndex = 0; otherIndex < *count; ++otherIndex) {
				if (index == otherIndex) {
					continue;
				}

				gravityX = positions[otherIndex].x - positions[index].x;
				gravityY = positions[otherIndex].y - positions[index].y;
				gravityZ = positions[otherIndex].z - positions[index].z;

				dist = sqrt(gravityX * gravityX + gravityY * gravityY + gravityZ * gravityZ);
				gravityX /= dist;
				gravityY /= dist;
				gravityZ /= dist;

				force = constGravity * (mass * masses[otherIndex]) / (dist * dist);
				gravityX *= force;
				gravityY *= force;
				gravityZ *= force;

				atomicAdd(&forces[index].x, gravityX);
				atomicAdd(&forces[index].y, gravityY);
				atomicAdd(&forces[index].z, gravityZ);
			}
		}
	}

	__global__ void UpdatePositionsGpu(unsigned int* count, unsigned int* offset, CUDA::Vector3* positions, CUDA::Vector3* velocities, float* masses, CUDA::Vector3* forces, float* dt) {
		int indexT = threadIdx.x + blockIdx.x * blockDim.x;
		int startIndex = indexT * *offset;
		int sizeIndex = startIndex + *offset;
		if (sizeIndex >= *count) {
			sizeIndex = *count;
		}

		float accelerationX;
		float accelerationY;
		float accelerationZ;
		float appendVelocityX;
		float appendVelocityY;
		float appendVelocityZ;

		for (int index = startIndex; index < sizeIndex; ++index) {
			accelerationX = forces[index].x / masses[index];
			accelerationY = forces[index].y / masses[index];
			accelerationZ = forces[index].z / masses[index];

			appendVelocityX = accelerationX * *dt;
			appendVelocityY = accelerationY * *dt;
			appendVelocityZ = accelerationZ * *dt;

			velocities[index].x += appendVelocityX;
			velocities[index].y += appendVelocityY;
			velocities[index].z += appendVelocityZ;

			positions[index].x += velocities[index].x * *dt;
			positions[index].y += velocities[index].y * *dt;
			positions[index].z += velocities[index].z * *dt;
		}
	}
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////

void WrapperX1::CalculateForceGPU(cuda::Buffer& buffer) {
	const unsigned int maxCountBlock = 65535;
	const unsigned int maxCountThread = CUDA::multithread ? CUDA::maxThreadsPerBlock : 1;

	unsigned int count = buffer.count;
	unsigned int countBlock;
	unsigned int countThread;
	unsigned int offset;

	CUDA::GetOffsets(count, maxCountBlock, maxCountThread, countBlock, countThread, offset);

	unsigned int* devCount;
	unsigned int* devOffset;
	CUDA::Vector3* devPositions;
	float* devMasses;
	CUDA::Vector3* devForces;

	hipMalloc(&devCount, sizeof(unsigned int));
	hipMalloc(&devOffset, sizeof(unsigned int));
	hipMalloc(&devPositions, count * sizeof(CUDA::Vector3));
	hipMalloc(&devMasses, count * sizeof(float));
	hipMalloc(&devForces, count * sizeof(CUDA::Vector3));

	hipMemcpy(devCount, &count, sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(devOffset, &offset, sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(devPositions, buffer.positions.data(), count * sizeof(CUDA::Vector3), hipMemcpyHostToDevice);
	hipMemcpy(devMasses, buffer.masses.data(), count * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(devForces, buffer.forces.data(), count * sizeof(CUDA::Vector3), hipMemcpyHostToDevice);

	if (sync) {
		CalcForcesGpuSync << <countBlock, countThread >> > (devCount, devOffset, devPositions, devMasses, devForces);
	}
	else {
		CalcForcesGpu << <countBlock, countThread >> > (devCount, devOffset, devPositions, devMasses, devForces);
	}

	hipMemcpy(buffer.forces.data(), devForces, count * sizeof(CUDA::Vector3), hipMemcpyDeviceToHost);

	hipFree(devCount);
	hipFree(devOffset);
	hipFree(devPositions);
	hipFree(devMasses);
	hipFree(devForces);
}

void WrapperX1::UpdatePositionGPU(cuda::Buffer& buffer, float dt) {
	const unsigned int maxCountBlock = 65535;
	const unsigned int maxCountThread = CUDA::multithread ? CUDA::maxThreadsPerBlock : 1;

	unsigned int count = buffer.count;
	unsigned int countBlock;
	unsigned int countThread;
	unsigned int offset;

	CUDA::GetOffsets(count, maxCountBlock, maxCountThread, countBlock, countThread, offset);

	unsigned int* devCount;
	unsigned int* devOffset;
	CUDA::Vector3* devPositions;
	float* devMasses;
	CUDA::Vector3* devForces;
	CUDA::Vector3* devVelocities;
	float* devDt;

	hipMalloc(&devCount, sizeof(unsigned int));
	hipMalloc(&devOffset, sizeof(unsigned int));
	hipMalloc(&devPositions, count * sizeof(CUDA::Vector3));
	hipMalloc(&devMasses, count * sizeof(float));
	hipMalloc(&devForces, count * sizeof(CUDA::Vector3));
	hipMalloc(&devVelocities, count * sizeof(CUDA::Vector3));
	hipMalloc(&devDt, sizeof(float));

	hipMemcpy(devCount, &count, sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(devOffset, &offset, sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(devPositions, buffer.positions.data(), count * sizeof(CUDA::Vector3), hipMemcpyHostToDevice);
	hipMemcpy(devMasses, buffer.masses.data(), count * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(devForces, buffer.forces.data(), count * sizeof(CUDA::Vector3), hipMemcpyHostToDevice);
	hipMemcpy(devVelocities, buffer.velocities.data(), count * sizeof(CUDA::Vector3), hipMemcpyHostToDevice);
	hipMemcpy(devDt, &dt, sizeof(float), hipMemcpyHostToDevice);

	UpdatePositionsGpu << <countBlock, countThread >> > (devCount, devOffset, devPositions, devVelocities, devMasses, devForces, devDt);

	hipMemcpy(buffer.positions.data(), devPositions, count * sizeof(CUDA::Vector3), hipMemcpyDeviceToHost);

	hipFree(devCount);
	hipFree(devOffset);
	hipFree(devPositions);
	hipFree(devMasses);
	hipFree(devForces);
	hipFree(devVelocities);
	hipFree(devDt);
}
