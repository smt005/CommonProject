#include "hip/hip_runtime.h"
#include "Test.h"

#if ENABLE_CUDA

#include <hip/hip_runtime.h>
#include <>

#include <vector>
#include <string>
#include <thread>
#include <stdio.h>

namespace {
	void fun0Cpu(int* count, int* arrayInt, int* arrayIntResult) {
		int val = 0;

		for (int i = 0; i < *count; ++i) {
			arrayIntResult[i] = arrayInt[i] * arrayInt[i];
		}
	}

	void fun1Cpu(int* count, int* arrayInt, int* sum) {
		*sum = 0;

		for (int i = 0; i < *count; ++i) {
			*sum += arrayInt[i];
		}
	}

	__global__
	void fun0Gpu(int* count, int* arrayInt, int* arrayIntResult) {
		int val = 0;

		for (int i = 0; i < *count; ++i) {
			arrayIntResult[i] = arrayInt[i] * arrayInt[i];
		}
	}

	__global__
	void fun1Gpu(int* count, int* arrayInt, int* sum) {
		*sum = 0;

		for (int i = 0; i < *count; ++i) {
			*sum += arrayInt[i];
		}
	}
}

void CUDA_Test::Run() {
	printf("Test::Run begin.\n");
	
	int count = 1000;
	std::vector<int> ints;
	ints.reserve(count);

	for (int i = 0; i < count; ++i) {
		ints.emplace_back(i);
	}

	// CPU
	int sumCPU;
	{
		std::vector<int> resultInts;
		resultInts.resize(count, 0);

		fun0Cpu(&count, ints.data(), resultInts.data());
		fun1Cpu(&count, resultInts.data(), &sumCPU);
	}

	// GPU
	int sumGPU;
	{
		int* devCount;
		int* devSum;
		int* devArrayInt;
		int* devArrayIntResult;

		std::vector<int> resultInts;
		resultInts.resize(count, 0);

		hipMalloc(&devCount,                  sizeof(int));
		hipMalloc(&devSum,                    sizeof(int));
		hipMalloc(&devArrayInt,       count * sizeof(int));
		hipMalloc(&devArrayIntResult, count * sizeof(int));

		hipMemcpy(devCount,         &count, sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(devSum,          &sumGPU, sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(devArrayInt, ints.data(), count * sizeof(int), hipMemcpyHostToDevice);

		fun0Gpu <<<1, 1>>> (devCount, devArrayInt, devArrayIntResult);

		hipMemcpy(&count, devCount, sizeof(int), hipMemcpyDeviceToHost);

		fun1Gpu <<<1, 1>>> (devCount, devArrayIntResult, devSum);

		hipMemcpy(&sumGPU,  devSum, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(resultInts.data(), devArrayIntResult, count * sizeof(int), hipMemcpyDeviceToHost);

		hipFree(devCount);
		hipFree(devSum);
		hipFree(devArrayInt);
		hipFree(devArrayIntResult);
	}

	//...
	if (sumCPU == sumGPU) {
		printf("Test::Run result OK [%i, %i].\n", sumCPU, sumGPU);
	}
	else {
		printf("Test::Run result FAIL [%i, %i].\n", sumCPU, sumGPU);
	}

	printf("Test::Run end.\n");
}

#else

	void CUDA_Test::Run() { }

#endif
