#include "hip/hip_runtime.h"
#include "Test.h"

#if ENABLE_CUDA

#include <hip/hip_runtime.h>
#include <>

#include <vector>
#include <string>
#include <functional>
#include <thread>
#include <stdio.h>
#include <algorithm>

#include "Wrapper.h"

namespace CUDA_TEST {
	// threadIdx.x
	struct ThreadIdx {
		unsigned int x = 0;
	} _threadIdx;

	// blockIdx.x
	struct BlockIdx {
		unsigned int x = 0;
	} _blockIdx;

	// blockDim.x
	struct BlockDim {
		unsigned int x = 0;
	} _blockDim;

	template<unsigned int countBlock, unsigned int countThread>
	void UmulateCuda(std::function<void(void)> fun) {
		_blockDim.x = countThread;

		for (unsigned int iBlock = 0; iBlock < countBlock; ++iBlock) {
			_blockIdx.x = iBlock;

			for (unsigned int iThread = 0; iThread < countThread; ++iThread) {
				_threadIdx.x = iThread;

				fun();
			}
		}			
	}

	void CalcForcesCpu(int* count, CUDA::Vector3* positions, float* masses, CUDA::Vector3* forces) {
		float constGravity = 0.01;
		int statIndex = 0;
		int countIndex = *count;

		float gravityX;
		float gravityY;
		float gravityZ;
		float dist = 0;
		float force = 0;

		// INFO
		printf("CalcForcesCpu statIndex: %i countIndex: %i\n", statIndex, countIndex);
		for (int index = statIndex; index < countIndex; ++index) {
			printf("CalcForcesCpu positions: [%f, %f, %f], masses: %f, forces: [%f, %f, %f]\n",
				positions[index].x, positions[index].y, positions[index].z,
				masses[index],
				forces[index].x, forces[index].y, forces[index].z);
		}

		for (int index = statIndex; index < countIndex; ++index) {
			CUDA::Vector3* pos = &positions[index];
			float mass = masses[index];
			forces[index].x = 0.f;
			forces[index].y = 0.f;
			forces[index].z = 0.f;

			for (size_t otherIndex = 0; otherIndex < *count; ++otherIndex) {
				if (index == otherIndex) {
					continue;
				}

				gravityX = positions[otherIndex].x - positions[index].x;
				gravityY = positions[otherIndex].y - positions[index].y;
				gravityZ = positions[otherIndex].z - positions[index].z;

				dist = sqrt(gravityX * gravityX + gravityY * gravityY + gravityZ * gravityZ);
				gravityX /= dist;
				gravityY /= dist;
				gravityZ /= dist;

				force = constGravity * (mass * masses[otherIndex]) / (dist * dist);
				gravityX *= force;
				gravityY *= force;
				gravityZ *= force;

				forces[index].x += gravityX;
				forces[index].y += gravityY;
				forces[index].z += gravityZ;
			}
		}
	}

	void UpdatePositionsCpu(int* count, CUDA::Vector3* positions, CUDA::Vector3* velocities, float* masses, CUDA::Vector3* forces, float* dt) {
		int statIndex = 0;
		int countIndex = *count;

		float accelerationX;
		float accelerationY;
		float accelerationZ;
		float appendVelocityX;
		float appendVelocityY;
		float appendVelocityZ;

		// INFO
		printf("UpdatePositionsCpu statIndex: %i countIndex: %i, dt: %f\n", statIndex, countIndex, *dt);
		for (int index = statIndex; index < countIndex; ++index) {
			printf("UpdatePositionsCpu positions: [%f, %f, %f], velocities: [%f, %f, %f], masses: %f, forces: [%f, %f, %f]\n",
				positions[index].x, positions[index].y, positions[index].z,
				velocities[index].x, velocities[index].y, velocities[index].z,
				masses[index],
				forces[index].x, forces[index].y, forces[index].z);
		}

		for (int index = statIndex; index < countIndex; ++index) {
			accelerationX = forces[index].x / masses[index];
			accelerationY = forces[index].y / masses[index];
			accelerationZ = forces[index].z / masses[index];

			appendVelocityX = accelerationX * *dt;
			appendVelocityY = accelerationY * *dt;
			appendVelocityZ = accelerationZ * *dt;

			velocities[index].x += appendVelocityX;
			velocities[index].y += appendVelocityY;
			velocities[index].z += appendVelocityZ;

			positions[index].x = velocities[index].x;
			positions[index].y = velocities[index].y;
			positions[index].z = velocities[index].z;
		}
	}

	__global__ void CalcForcesGpu(int* count, CUDA::Vector3* positions, float* masses, CUDA::Vector3* forces) {
		float constGravity = 0.01;
		int statIndex = 0;
		int countIndex = *count;

		float gravityX;
		float gravityY;
		float gravityZ;
		float dist = 0;
		float force = 0;

		// INFO
		printf("CalcForcesGpu statIndex: %i countIndex: %i\n", statIndex, countIndex);
		for (int index = statIndex; index < countIndex; ++index) {
			printf("CalcForcesCpu positions: [%f, %f, %f], masses: %f, forces: [%f, %f, %f]\n",
				positions[index].x, positions[index].y, positions[index].z,
				masses[index],
				forces[index].x, forces[index].y, forces[index].z);
		}

		for (int index = statIndex; index < countIndex; ++index) {
			CUDA::Vector3* pos = &positions[index];
			float mass = masses[index];
			forces[index].x = 0.f;
			forces[index].y = 0.f;
			forces[index].z = 0.f;

			for (size_t otherIndex = 0; otherIndex < *count; ++otherIndex) {
				if (index == otherIndex) {
					continue;
				}

				gravityX = positions[otherIndex].x - positions[index].x;
				gravityY = positions[otherIndex].y - positions[index].y;
				gravityZ = positions[otherIndex].z - positions[index].z;

				dist = sqrt(gravityX * gravityX + gravityY * gravityY + gravityZ * gravityZ);
				gravityX /= dist;
				gravityY /= dist;
				gravityZ /= dist;

				force = constGravity * (mass * masses[otherIndex]) / (dist * dist);
				gravityX *= force;
				gravityY *= force;
				gravityZ *= force;

				forces[index].x += gravityX;
				forces[index].y += gravityY;
				forces[index].z += gravityZ;
			}
		}
	}

	__global__ void UpdatePositionsGpu(int* count, CUDA::Vector3* positions, CUDA::Vector3* velocities, float* masses, CUDA::Vector3* forces, float* dt) {
		int statIndex = 0;
		int countIndex = *count;

		float accelerationX;
		float accelerationY;
		float accelerationZ;
		float appendVelocityX;
		float appendVelocityY;
		float appendVelocityZ;

		// INFO
		printf("UpdatePositionsGpu statIndex: %i countIndex: %i, dt: %f\n", statIndex, countIndex, *dt);
		for (int index = statIndex; index < countIndex; ++index) {
			printf("UpdatePositionsCpu positions: [%f, %f, %f], velocities: [%f, %f, %f], masses: %f, forces: [%f, %f, %f]\n",
				positions[index].x, positions[index].y, positions[index].z,
				velocities[index].x, velocities[index].y, velocities[index].z,
				masses[index],
				forces[index].x, forces[index].y, forces[index].z);
		}

		for (int index = statIndex; index < countIndex; ++index) {
			accelerationX = forces[index].x / masses[index];
			accelerationY = forces[index].y / masses[index];
			accelerationZ = forces[index].z / masses[index];

			appendVelocityX = accelerationX * *dt;
			appendVelocityY = accelerationY * *dt;
			appendVelocityZ = accelerationZ * *dt;

			velocities[index].x += appendVelocityX;
			velocities[index].y += appendVelocityY;
			velocities[index].z += appendVelocityZ;

			positions[index].x = velocities[index].x;
			positions[index].y = velocities[index].y;
			positions[index].z = velocities[index].z;
		}
	}
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// CUDA_Test::Run //////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void CUDA_Test::Run() {
	int count = 10;

	printf("Test::Run begin count: %i.\n", count);
	
	static std::vector<CUDA::Vector3> positions;
	std::vector<float> masses;
	std::vector<CUDA::Vector3> velocities;
	float dt = 1.f;

	if (positions.empty()) {
		positions.reserve(count);
		masses.reserve(count);
		velocities.reserve(count);

		for (int i = 0; i < count; ++i) {
			positions.emplace_back((float)i, (float)i, (float)i);
			masses.emplace_back((float)i + 1.f);
			velocities.emplace_back((float)i, (float)i, (float)i);
		}
	}

	// CPU
	std::vector<CUDA::Vector3> cpuPositions;
	{
		printf("Test::Run CPU begin\n");
		cpuPositions = positions;
		std::vector<CUDA::Vector3> cpuVelocities = velocities;

		std::vector<CUDA::Vector3> forces;
		forces.resize(count, CUDA::Vector3());

		//CalcForcesCpu(&count, cpuPositions.data(), masses.data(), forces.data());
		//UpdatePositionsCpu(&count, cpuPositions.data(), cpuVelocities.data(), masses.data(), forces.data(), &dt);
		
		CUDA_TEST::UmulateCuda<1, 1>([&count, &cpuPositions, &masses, &forces]() { CUDA_TEST::CalcForcesCpu(&count, cpuPositions.data(), masses.data(), forces.data()); });
		CUDA_TEST::UmulateCuda<1, 1>([&count, &cpuPositions, &cpuVelocities, &masses, &forces, &dt]() { CUDA_TEST::UpdatePositionsCpu(&count, cpuPositions.data(), cpuVelocities.data(), masses.data(), forces.data(), &dt); });

		printf("Test::Run CPU end\n\n");
	}

	// GPU
	std::vector<CUDA::Vector3> gpuPositions;
	{
		printf("Test::Run GPU begin\n");

		gpuPositions.resize(count);
		std::vector<CUDA::Vector3> gpuVelocities = velocities;

		int* devCount;
		float* devDt;
		CUDA::Vector3* devPositions;
		float* devMasses;
		CUDA::Vector3* devVelocities;
		CUDA::Vector3* devForces;

		hipMalloc(&devCount,	sizeof(int));
		hipMalloc(&devDt,	sizeof(float));
		hipMalloc(&devPositions,	count * sizeof(CUDA::Vector3));
		hipMalloc(&devMasses,	count * sizeof(float));
		hipMalloc(&devVelocities,	count * sizeof(CUDA::Vector3));
		hipMalloc(&devForces, count * sizeof(CUDA::Vector3));

		hipMemcpy(devCount,	&count, sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(devDt,	&dt, sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(devPositions, positions.data(), count * sizeof(CUDA::Vector3), hipMemcpyHostToDevice);
		hipMemcpy(devMasses, masses.data(), count * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(devVelocities, gpuVelocities.data(), count * sizeof(CUDA::Vector3), hipMemcpyHostToDevice);

		CUDA_TEST::CalcForcesGpu << <1, 1 >> > (devCount, devPositions, devMasses, devForces);
		CUDA_TEST::UpdatePositionsGpu << <1, 1 >> > (devCount, devPositions, devVelocities, devMasses, devForces, devDt);

		hipMemcpy(gpuPositions.data(), devPositions, count * sizeof(CUDA::Vector3), hipMemcpyDeviceToHost);
		hipMemcpy(gpuVelocities.data(), devVelocities, count * sizeof(CUDA::Vector3), hipMemcpyDeviceToHost);

		hipFree(devCount);
		hipFree(devDt);
		hipFree(devPositions);
		hipFree(devMasses);
		hipFree(devForces);
		hipFree(devVelocities);

		printf("Test::Run GPU end\n\n");
	}

	printf("Test::Run end.\n\n");

	//...
	bool equal = true;
	for (size_t i = 0; i < count; ++i) {
		if (!(cpuPositions[i].x == gpuPositions[i].x, cpuPositions[i].y == gpuPositions[i].y, cpuPositions[i].z == gpuPositions[i].z)) {
			equal = false;
			break;
		}
	}
	
	if (equal) {
		printf("Test::Run result OK.\n");
	}
	else {
		printf("Test::Run result FAIL.\n\n");

		for (size_t i = 0; i < count; ++i) {
			printf("\tpos: [%f, %f, %f] != [%f, %f, %f]\n", cpuPositions[i].x, cpuPositions[i].y, cpuPositions[i].z, gpuPositions[i].x, gpuPositions[i].y, gpuPositions[i].z);
		}
	}
}


/////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// RunTestIndex ////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////

namespace CUDA_TEST {

void TestIndexCpu(int* count, int* indexes, int* result) {
	int index = _threadIdx.x + _blockIdx.x * _blockDim.x;

	if (index < *count) {
		indexes[index] = index;
		*result += index;

		printf("TestIndexCpu APPEND index: %i = %i + (%i * %i)\n", index, _threadIdx.x, _blockIdx.x, _blockDim.x);
	}
	else {
		printf("TestIndexCpu  skip  index: %i = %i + (%i * %i)\n", index, _threadIdx.x, _blockIdx.x, _blockDim.x);
	}
}

__global__ void TestIndexGpu(int* count, int* indexes, int* result) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	
	if (index < *count) {
		indexes[index] = index;
		*result += index;

		printf("TestIndexGpu APPEND index: %i = %i + (%i * %i)\n", index, threadIdx.x, blockIdx.x, blockDim.x);
	} else {
		printf("TestIndexGpu  skip  index: %i = %i + (%i * %i)\n", index, threadIdx.x, blockIdx.x, blockDim.x);
	}
}

// FOR
void TestIndexForCpu(int* count, int* offset, int* indexes, int* result) {
	int indexT = _threadIdx.x + _blockIdx.x * _blockDim.x;
	printf("\nTestIndexForCpu APPEND index: %i (%i, %i, %i)\n", indexT, _threadIdx.x, _blockIdx.x, _blockDim.x);

	int startIndex = indexT * *offset;
	int sizeIndex = startIndex + *offset;
	if (sizeIndex >= *count) {
		sizeIndex = *count;
		printf("TestIndexForCpu CORRECT [%i, %i]\n", sizeIndex, *count);
	}

	for (int index = startIndex; index < sizeIndex; ++index) {
		indexes[index] = index;
		*result += index;

		printf("TestIndexForCpu [%i] APPEND index: %i\n", indexT, index);
	}
}

__global__ void TestIndexForGpu(int* count, int* offset, int* indexes, int* result) {
	int indexT = threadIdx.x + blockIdx.x * blockDim.x;
	printf("\nTestIndexForGpu APPEND index: %i (%i, %i, %i)\n", indexT, threadIdx.x, blockIdx.x, blockDim.x);

	int startIndex = indexT * *offset;
	int sizeIndex = startIndex + *offset;
	if (sizeIndex >= *count) {
		sizeIndex = *count;
		printf("TestIndexForGpu CORRECT [%i, %i]\n", sizeIndex, *count);
	}

	for (int index = startIndex; index < sizeIndex; ++index) {
		indexes[index] = index;

		//__syncthreads();
		*result += index;

		printf("TestIndexForGpu [%i] APPEND index: %i\n", indexT, index);
	}
}

}

void CUDA_Test::RunTestIndex() {
	printf("\nTest::RunTestIndex BEGIN.\n");

	constexpr int count = 33;
	int reserveCount = count;
	constexpr int maxCountBlock = 2;
	constexpr int maxCountThread = 2;

	constexpr int countThread = maxCountThread;
	constexpr int countBlock = ((count + countThread - 1) / countThread) > maxCountBlock ? maxCountBlock : ((count + countThread - 1) / countThread);
	constexpr int offset = (count + (countBlock * countThread) - 1) / (countBlock * countThread);

	bool _for_ = true;

	// CPU
	int resultCpu = 0;
	std::vector<int> indexesCpu;
	{
		printf("\nCPU .   .   .\n");

		indexesCpu.resize(reserveCount, std::numeric_limits<int>::max());

		int* devCount = new int(count);
		int* devOffset = new int(offset);

		if (_for_) {
			CUDA_TEST::UmulateCuda<countBlock, countThread>([devCount, devOffset, devResult = &resultCpu, devIndexes = indexesCpu.data()]() {
				CUDA_TEST::TestIndexForCpu(devCount, devOffset, devIndexes, devResult);
			});
		} else {
			CUDA_TEST::UmulateCuda<countBlock, countThread>([devCount, devResult = &resultCpu, devIndexes = indexesCpu.data()]() {
				CUDA_TEST::TestIndexCpu(devCount, devIndexes, devResult);
			});
		}

		delete devCount;
		delete devOffset;
	}

	// GPU
	int resultGpu = 0;
	std::vector<int> indexesGpu;
	{
		printf("\nGPU .   .   .\n");
		
		indexesGpu.resize(reserveCount, std::numeric_limits<int>::max());

		int* devCount;
		int* devOffset;
		int* devResult;
		int* devIndexes;

		hipMalloc(&devCount, sizeof(int));
		hipMalloc(&devOffset, sizeof(int));
		hipMalloc(&devResult, sizeof(int));
		hipMalloc(&devIndexes, reserveCount * sizeof(int));

		hipMemcpy(devCount, &count, sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(devOffset, &offset, sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(devResult, &resultGpu, sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(devIndexes, indexesGpu.data(), reserveCount * sizeof(int), hipMemcpyHostToDevice);

		if (_for_) {
			CUDA_TEST::TestIndexForGpu << <countBlock, countThread >> > (devCount, devOffset, devIndexes, devResult);
		} else {
			CUDA_TEST::TestIndexGpu << <countBlock, countThread >> > (devCount, devIndexes, devResult);
		}

		hipMemcpy(&resultGpu, devResult, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(indexesGpu.data(), devIndexes, reserveCount * sizeof(int), hipMemcpyDeviceToHost);

		hipFree(devCount);
		hipFree(devOffset);
		hipFree(devResult);
		hipFree(devIndexes);
	}

	printf("\nTest::RunTestIndex END.\n");

	if (resultCpu == resultGpu) {
		printf("Test::RunTestIndex result [%i, %i] OK.\n", resultCpu, resultGpu);
	}
	else {
		printf("Test::RunTestIndex result [%i, %i] FAIL.\n", resultCpu, resultGpu);
	}

	std::sort(indexesCpu.begin(), indexesCpu.end());
	std::sort(indexesGpu.begin(), indexesGpu.end());

	for (size_t i = 0; i < reserveCount; ++i) {
		printf("\tindex: %i: [%i, %i]\n", i, indexesCpu[i], indexesGpu[i]);
	}
}

#else

	void CUDA_Test::Run() { }
	void CUDA_Test::RunTestIndex() { }

#endif
