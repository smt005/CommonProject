#include "hip/hip_runtime.h"

#include  "WrapperPrototypeV3.h"
#include <thread>
#include <vector>
#include <iostream>

#if ENABLE_CUDA

#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <string>

//////////////////////////////////////////////////////////////////////////////////////////////////////////
// GPU ///////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////
    
namespace {
    __global__
        void GetForceForGPU(int* count, int* offset, CUDA::Body* bodies, CUDA::Vector3* forces) {
        float _constGravity = 0.01f;
        int statIndex = 0;// threadIdx.x;// +blockIdx.x * blockDim.x;
        int endIndex = *count;// statIndex + *offset;
        if (endIndex > *count) {
            endIndex = *count;
        }

        int sizeData = *count;
        float gravityVecX;
        float gravityVecY;
        float gravityVecZ;
        float dist;
        float force;

        for (int index = statIndex; index < endIndex; ++index) {
            for (size_t otherIndex = 0; otherIndex < sizeData; ++otherIndex) {
                if (index == otherIndex) {
                    continue;
                }

                gravityVecX = bodies[otherIndex].pos.x - bodies[index].pos.x;
                gravityVecY = bodies[otherIndex].pos.y - bodies[index].pos.y;
                gravityVecZ = bodies[otherIndex].pos.z - bodies[index].pos.z;

                dist = sqrt(gravityVecX * gravityVecX + gravityVecY * gravityVecY + gravityVecZ * gravityVecZ);
                gravityVecX /= dist;
                gravityVecY /= dist;
                gravityVecZ /= dist;

                force = _constGravity * (bodies[index].mass * bodies[otherIndex].mass) / (dist * dist);
                gravityVecX *= force;
                gravityVecY *= force;
                gravityVecZ *= force;

                forces[index].x += gravityVecX;
                forces[index].y += gravityVecY;
                forces[index].z += gravityVecZ;
            }
        }
    }
    
    __global__
        void GetForceGPU(int* count, CUDA::Body* bodyes, CUDA::Vector3* forces, int* devImin, int* devImax) {
        int index = threadIdx.x;// +blockIdx.x * blockDim.x;
        if (index >= *count) {
            return;
        }

        if (index < *devImin) {
            *devImin = index;
        }
        if (index > *devImax) {
            *devImax = index;
        }

        double _constGravity = 0.01f;
        int sizeData = *count;
        float gravityVecX = 0;
        float gravityVecY = 0;
        float gravityVecZ = 0;
        float dist;
        float force;
        float mass = bodyes[index].mass;

        float posX = bodyes[index].pos.x;
        float posY = bodyes[index].pos.y;
        float posZ = bodyes[index].pos.z;

        for (int otherIndex = 0; otherIndex < sizeData; ++otherIndex) {
            if (index == otherIndex) {
                continue;
            }

            gravityVecX = bodyes[otherIndex].pos.x - posX;
            gravityVecY = bodyes[otherIndex].pos.y - posY;
            gravityVecZ = bodyes[otherIndex].pos.z - posZ;

            dist = sqrt(gravityVecX * gravityVecX + gravityVecY * gravityVecY + gravityVecZ * gravityVecZ);
            gravityVecX /= dist;
            gravityVecY /= dist;
            gravityVecZ /= dist;

            force = _constGravity * (mass * bodyes[otherIndex].mass) / (dist * dist);
            gravityVecX *= force;
            gravityVecY *= force;
            gravityVecZ *= force;

            forces[index].x += gravityVecX;
            forces[index].y += gravityVecY;
            forces[index].z += gravityVecZ;
        }
    }
}

int CUDA_PrototypeV3::tag = 0;
int CUDA_PrototypeV3::tagCurrent = -1;

void CUDA_PrototypeV3::GetForcesGPUStatic(std::vector<CUDA::Body>& bodies, std::vector<CUDA::Vector3>& forces) {
    int count = bodies.size();
    forces.resize(count, CUDA::Vector3());

    unsigned int counThread = count < CUDA::maxThreadsPerBlock ? count : CUDA::maxThreadsPerBlock;

    unsigned int countBlock = (count + counThread - 1) / counThread;
    countBlock = countBlock > CUDA::maxGridSize[1] ? CUDA::maxGridSize[1] : countBlock;

    int offset = count / (counThread * countBlock);
    if ((count % (counThread * countBlock)) > 0) {
        ++offset;
    }

    //...
    int* devCount;
    int* devOffset;
    CUDA::Body* devBodyes;
    CUDA::Vector3* devForces;

    hipError_t error;

    hipMalloc(&devCount,               sizeof(int));
    hipMalloc(&devOffset,              sizeof(int));
    hipMalloc(&devBodyes,      count * sizeof(CUDA::Body));
    hipMalloc(&devForces,      count * sizeof(CUDA::Vector3));

    int iMin = 10000;
    int iMax = -10000;
    int* devImin;
    int* devImax;

    hipMalloc(&devImin, sizeof(int));
    hipMalloc(&devImax, sizeof(int));

    error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA_PrototypeV3 hipMalloc error: %s\n", hipGetErrorString(error));
        return;
    }

    hipMemcpy(devCount,        &count,              sizeof(int),           hipMemcpyHostToDevice);
    hipMemcpy(devOffset,       &offset,             sizeof(int),           hipMemcpyHostToDevice);
    hipMemcpy(devBodyes, bodies.data(),     count * sizeof(CUDA::Body), hipMemcpyHostToDevice);
    hipMemcpy(devForces, bodies.data(),    count * sizeof(CUDA::Vector3), hipMemcpyHostToDevice);

    hipMemcpy(devImin, &iMin, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(devImax, &iMax, sizeof(int), hipMemcpyHostToDevice);

    error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA_PrototypeV3 hipMemcpy error: %s\n", hipGetErrorString(error));
        return;
    }

    if (tag == 0) {
        if (tag != tagCurrent) {
            tagCurrent = tag;
            printf("GPU: [%i] CUDA_PrototypeV3::GetForceForGPU\n", tagCurrent);
        }
        //GetForceForGPU <<<countBlock, counThread>>> (devCount, devOffset, devBodyes, devForces);
        GetForceForGPU << <1, 1 >> > (devCount, devOffset, devBodyes, devForces);

        error = hipGetLastError();
        if (error != hipSuccess) {
            printf("CUDA_PrototypeV3 GetForceForGPU error: %s\n", hipGetErrorString(error));
            return;
        }
    }
    else if (tag == 1) {
        if (tag != tagCurrent) {
            tagCurrent = tag;
            printf("GPU: [%i] CUDA_PrototypeV3::GetForceGPU\n", tagCurrent);
        }
        //GetForceGPU <<<countBlock, counThread>>> (devCount, devBodyes, devForces);
        GetForceGPU << <1, count >> > (devCount, devBodyes, devForces, devImin, devImax);

        error = hipGetLastError();
        if (error != hipSuccess) {
            printf("CUDA_PrototypeV3 GetForceGPU error: %s\n", hipGetErrorString(error));
            return;
        }
    }

    hipMemcpy(forces.data(), devForces, count * sizeof(CUDA::Vector3), hipMemcpyDeviceToHost);

    hipMemcpy(&iMin, devImin, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&iMax, devImax, sizeof(int), hipMemcpyDeviceToHost);

    if (iMin < 0 || iMax >= count) {
        printf("CUDA_PrototypeV3 error count: [%i, %i]\n", iMin, iMax);
    }

    error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA_PrototypeV3 hipMemcpy error: %s\n", hipGetErrorString(error));
        return;
    }

    hipFree(devCount);
    hipFree(devOffset);
    hipFree(devBodyes);
    hipFree(devForces);

    hipFree(devImin);
    hipFree(devImax);

    error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA_PrototypeV3 hipFree error: %s\n", hipGetErrorString(error));
        return;
    }
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////
// CPU ///////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////

namespace {
    void GetForceCPU(int* count, int* offset, CUDA::Body* bodies, CUDA::Vector3* forces, int threadId) {
        float _constGravity = 0.01f;
        int statIndex = 0;// threadIdx.x;// +blockIdx.x * blockDim.x;
        int endIndex = *count;// statIndex + *offset;
        if (endIndex > *count) {
            endIndex = *count;
        }

        int sizeData = *count;
        float gravityVecX;
        float gravityVecY;
        float gravityVecZ;
        float dist;
        float force;

        for (int index = statIndex; index < endIndex; ++index) {
            for (size_t otherIndex = 0; otherIndex < sizeData; ++otherIndex) {
                if (index == otherIndex) {
                    continue;
                }

                gravityVecX = bodies[otherIndex].pos.x - bodies[index].pos.x;
                gravityVecY = bodies[otherIndex].pos.y - bodies[index].pos.y;
                gravityVecZ = bodies[otherIndex].pos.z - bodies[index].pos.z;

                dist = sqrt(gravityVecX * gravityVecX + gravityVecY * gravityVecY + gravityVecZ * gravityVecZ);
                gravityVecX /= dist;
                gravityVecY /= dist;
                gravityVecZ /= dist;

                force = _constGravity * (bodies[index].mass * bodies[otherIndex].mass) / (dist * dist);
                gravityVecX *= force;
                gravityVecY *= force;
                gravityVecZ *= force;

                forces[index].x += gravityVecX;
                forces[index].y += gravityVecY;
                forces[index].z += gravityVecZ;
            }
        }
    }
}

void CUDA_PrototypeV3::GetForcesCPUStatic(std::vector<CUDA::Body>& bodies, std::vector<CUDA::Vector3>& forces) {
    int counThread = static_cast<double>(std::thread::hardware_concurrency());
    int count = bodies.size();
    forces.resize(count, CUDA::Vector3());

    /*if ((count * 2) > counThread) {
        int offst = count / counThread;
        if ((count % counThread) > 0) {
            ++offst;
        }

        std::vector<std::thread> threads;
        threads.reserve(counThread);

        for (int threadId = 0; threadId < counThread; ++threadId) {
            threads.emplace_back([&]() {
                GetForceCPU(&count, &offst, bodies.data(), forces.data(), threadId);
            });
        }

        for (std::thread& th : threads) {
            th.join();
        }
    }
    else*/
    {
        GetForceCPU(&count, &count, bodies.data(), forces.data(), 0);
    }
}

//...
#else
    void testCUDA(void) {}

    void CUDA::GetProperty() {}
    void CUDA::PrintInfo() {}
#endif
